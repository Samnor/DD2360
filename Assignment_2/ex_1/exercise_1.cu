
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 256
#define TPB 256

__global__ void cuda_hello(){
    printf("Hello World! My threadId is %d\n", threadIdx.x);
}

int main() {
    cuda_hello<<<N/TPB,TPB>>>();
    hipDeviceSynchronize();
    return 0;
}
